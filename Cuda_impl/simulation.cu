// Time is measured in frames


#include <hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<omp.h>
#include<stdbool.h>
#include<stdlib.h>
#include<math.h>
#include<stdio.h>
#include<hiprand/hiprand_kernel.h>

#define CUDA_CORES 384 //number of cuda cores

#define RAND_SEED 1999
#define POP_SIZE 100000
//#define POP_SIZE 2000
#define INIT_PROB_INFECTED 100 // per 10000
#define INIT_PROB_QUARANTINED 0
//#define R_CONTAGION 1.
#define R_CONTAGION 4.
#define P_CONTAGION 0.1  // x100 to get percentage
#define NDAYS_MAX 30
//#define NFRAMES_PER_DAY 100
#define NFRAMES_PER_DAY 100
#define DURATION_EXPOSURE_DAYS 2.9
#define DURATION_INFECTION_DAYS 6.4

#define CITY_WIDTH_METERS 1000


FILE *file;


typedef struct
{
  double speed;
  
  //current position 
  double pos_x;
  double pos_y;

  // target position
  double obj_x;
  double obj_y;

  //displacement per frame
  double delta_x;
  double delta_y;

  // state
  bool susceptible;
  bool exposed;
  bool infected;
  bool removed;
  bool quarantined;
  
  // frame number of event
  int t_exposed;
  int t_infected;
  int t_removed;

  // duration of exposed and infection period
  int duration_exposure;
  int duration_infection;

  int people_infected;
} Person;

// __global__ void expose(Person *person, int iframe)
// {
//     person->susceptible= false;
//     person->exposed = true;
//     person->infected = false;
//     person->removed = false;
//     person->t_exposed = iframe;
//     //    printf("expose: New exposed person\n");
//     (*num_exposed) ++;
//     (*num_susceptible) --;
// }

void infect(Person *person, int iframe, int *num_infected, int *num_exposed)
{
    person->susceptible = false;
    person->exposed = false;
    person->infected = true;
    person->removed = false;
    person->t_infected = iframe;
    //    printf("infect: New infectious person\n");
    (*num_infected)++;
    (*num_exposed)--;
}

void remove_p(Person *person, int iframe, int *num_removed, int *num_infected)
{
    person->susceptible = false;
    person->exposed = false;
    person->infected = false;
    person->removed = true;
    person->t_removed = iframe;
    //    printf("remove_p: New declared case\n");
    (*num_removed)++;
    (*num_infected)--;
}

void check_contamination(Person *person, int iframe, int *num_removed, int *num_infected, int *num_exposed)
{
  if ((person->exposed) && (iframe - person->t_exposed >= person->duration_exposure)) {
    infect(person, iframe, num_infected, num_exposed);
  }
  if ((person->infected) && (iframe - person->t_infected >= person->duration_infection)) {
    remove_p(person, iframe, num_removed, num_infected);
  }
}


void set_objective(Person *person, int obj_x, int obj_y)
{
    person->obj_x = obj_x;
    person->obj_y = obj_y;
    if(person->quarantined)
    {
        person->delta_x = 0;
        person->delta_y = 0;
    }
    else
    {
        person->delta_x = ( person->obj_x - person->pos_x ) / person->speed;
        person->delta_y = ( person->obj_y - person->pos_y ) / person->speed;
    }
}

void update_pos(Person *person, double n_pos_x, double n_pos_y)
{
  if(n_pos_x == 0 && n_pos_y == 0)
    {
      person->pos_x = person->pos_x + person->delta_x;
      person->pos_y = person->pos_y + person->delta_y;
    }
  else
    {
      person->pos_x = n_pos_x;
      person->pos_y = n_pos_y;
    }
}

__device__ double get_distance(Person *p1, Person *p2)
{
  return sqrtf(powf(p1->pos_x - p2->pos_x, 2) + pow(p1->pos_y - p2->pos_y, 2));
}

int random_bounded_num(int lower, int upper)
{
    return (rand() % (upper - lower + 1)) + lower; 
}

__global__ void setup_kernel(hiprandState *state, int* max_i_per_core)
{
    int id = threadIdx.x;
    if(id < POP_SIZE)
        hiprand_init(RAND_SEED, id, 0, &state[id]);
        if (id == CUDA_CORES - 1)
        {
          max_i_per_core[id] = POP_SIZE - 1;
        }
        else
        {
          max_i_per_core[id] = (id + 1) * (POP_SIZE / CUDA_CORES);
        }
}

void generate_population(Person *population, int *num_exposed, int *num_susceptible)
{
  //#pragma omp parallel for
  int contaminated = 0;
  
  for(int ipop=0; ipop < POP_SIZE; ipop++)
    {
        Person p;
        p.pos_x = (double)random_bounded_num(0,CITY_WIDTH_METERS);
        p.pos_y = (double)random_bounded_num(0,CITY_WIDTH_METERS);

	p.duration_exposure = DURATION_EXPOSURE_DAYS * NFRAMES_PER_DAY;
	p.duration_infection = DURATION_INFECTION_DAYS * NFRAMES_PER_DAY;

        p.susceptible = true;
        p.exposed = false;
        p.infected = false;
        p.removed = false;

	p.t_exposed = -1;
	p.t_infected = -1;p.t_removed = -1;

	p.people_infected = 0;

    if (random_bounded_num(0,10000) < INIT_PROB_INFECTED)
    {
        p.susceptible= false;
        p.exposed = true;
        p.infected = false;
        p.removed = false;
        p.t_exposed = 0;
        (*num_exposed) ++;
        (*num_susceptible) --;
        contaminated++;
    }
	
	if (random_bounded_num(0,100) < INIT_PROB_QUARANTINED) p.quarantined = true;

        population[ipop] = p;
    }  // for ipop
    printf("generate_population: %d people are initially contaminated\n", contaminated);
}

void update_daily_objectives(Person *population)
{
  Person *p;

  for(int ipop=0; ipop < POP_SIZE; ipop++) {
    p = &(population[ipop]);
    if (!(p->removed)) {
	p->obj_x = (double)random_bounded_num(0,CITY_WIDTH_METERS);
	p->obj_y = (double)random_bounded_num(0,CITY_WIDTH_METERS);
	p->speed = sqrt(pow(p->obj_x-p->pos_x,2.)+pow(p->obj_y-p->pos_y,2.))/
	  ((double)NFRAMES_PER_DAY);
	set_objective(p, p->obj_x, p->obj_y);
      } // p->removed
  } // for ipop

}

__global__ void frame_update_helper(Person *person, Person *population, hiprandState *states, int iframe, int *num_exposed, int *num_susceptible, int* max_i_per_core)
{
    int i = threadIdx.x;
    int lower_bound, upper_bound;
    if(i == 0)
    {
      lower_bound = 0;
      upper_bound = max_i_per_core[i];
    }
    else
    {
      lower_bound = max_i_per_core[i-1];
      upper_bound = max_i_per_core[i];
    }


    if (i < POP_SIZE)
    {
      // printf("Id: %i, lower bound: %i, upper bound: %i\n", i, lower_bound, upper_bound);
      hiprandState localState = states[i];
      for (int j = lower_bound; j < upper_bound; j++)
      {
        float random_num = (hiprand_uniform(&localState));
        // printf("randnum : %f\n", random_num);
        if((person != &population[j]) 
        &&  population[j].susceptible 
        && (get_distance(person, &population[j]) < R_CONTAGION) 
        && ( random_num < P_CONTAGION))
        {
            population[j].susceptible = false;
            population[j].exposed = true;
            population[j].infected = false;
            population[j].removed = false;
            population[j].t_exposed = iframe;
            (*num_exposed) ++;
            (*num_susceptible) --;
            person->people_infected ++;
            // printf("I get here");
        }
      }
      states[i] = localState;
    }

}

void frame_update(int iframe,Person* population, hiprandState *states, int *removed, int *num_susceptible, int *num_exposed, int *num_infected, int* num_removed, int* max_i_per_core)
{
    for (int ipop = 0; ipop < POP_SIZE; ipop++) {
      if (!(population[ipop].removed)) {
	check_contamination(&population[ipop],iframe, num_removed, num_infected, num_exposed);
	update_pos(&population[ipop], 0, 0);
	
	//      printf("frame_update: iframe = %d, ipop = %d, removed = % d, t_removed = %d\n",
	//	     iframe, ipop, population[ipop].removed, population[ipop].t_removed);
	
	if ((population[ipop].removed) && (population[ipop].t_removed == iframe))
	  (*removed)++;
	
	if(population[ipop].infected) {
    frame_update_helper<<<1,CUDA_CORES>>>(&population[ipop], population, states, iframe, num_exposed, num_susceptible, max_i_per_core);
    hipDeviceSynchronize();
	} // if populuation[ipop]
      } // if populuation[ipop]
    } // for ipop
    //    printf("frame_update: new reported cases = %d\n", *removed);
}


double compute_R(int iday, Person *population)
{
  div_t ratio;
  int ninfected = 0;
  int ninfectors = 0;

  for (int ipop = 0; ipop < POP_SIZE; ipop++) {
    if (population[ipop].removed) {
	ratio = div(population[ipop].t_removed, NFRAMES_PER_DAY);
	if (ratio.quot == iday) {
	  //	  printf("compute_R: ipop = %d was removed on day = %d, having infected %d others\n", ipop, ratio.quot,
	  //		 population[ipop].people_infected); 
	  ninfected += population[ipop].people_infected;
	  ninfectors++;
	} // if ratio-quot
    } // if population[ipop]
  } // for ipop

  double repro_rate = 0.;
  if (ninfectors != 0) repro_rate = ((double)ninfected)/((double)ninfectors);

  return repro_rate;
}


int main()
{
  clock_t begin = clock();
  int *num_susceptible, *num_exposed, *num_infected, *num_removed;
  hipMallocManaged(&num_susceptible, sizeof(int), hipMemAttachGlobal);
  hipMallocManaged(&num_exposed, sizeof(int), hipMemAttachGlobal);
  hipMallocManaged(&num_infected, sizeof(int), hipMemAttachGlobal);
  hipMallocManaged(&num_removed, sizeof(int), hipMemAttachGlobal);
  *num_susceptible = POP_SIZE;
  *num_exposed = 0;
  *num_infected = 0;
  *num_removed = 0;

  // printf("susceptible = %i, exposed  = %i, infected = %i, removed = %i\n",
  // *num_exposed, *num_infected, *num_removed);

  file = fopen("output.dat", "w");
  if(file == NULL)
  {
    printf("Error with file\n");
    exit(1);
  }

  srand(RAND_SEED);

  Person *population;
  hipMallocManaged(&population, POP_SIZE * sizeof(Person), hipMemAttachGlobal);

  hiprandState *devStates;
  hipMalloc((void **)&devStates, CUDA_CORES * sizeof(hiprandState));

  int *max_i_per_core;
  hipMalloc ((void **) &max_i_per_core, CUDA_CORES * sizeof(int));

  setup_kernel<<<1, CUDA_CORES>>>(devStates, max_i_per_core);
  
  generate_population(population, num_exposed, num_susceptible);
  int iframe = 0;
  int iday = 0;
  int total_removed = 0;
  
  fprintf(file, "#frame\tday\tnewreported cases\ttotal removed\tR\n");
  printf("#frame\t day\t new reported cases\t total removed\t R\n");
  while((total_removed < POP_SIZE) && (iframe < NDAYS_MAX*NFRAMES_PER_DAY)) {
    int removed = 0;
    update_daily_objectives(population);
    for (int iframeperday=0; iframeperday < NFRAMES_PER_DAY; iframeperday++) {
      frame_update(iframe, population, devStates,  &removed, num_susceptible, num_exposed, num_infected, num_removed, max_i_per_core);
      printf("Time frame %d: susceptible = %d, exposed  = %d, infected = %d, removed = %d, with %d new reported cases\n",
	          iframe, *num_susceptible,
	          *num_exposed, *num_infected, *num_removed, removed);
      iframe++;
    } // for iframeperday
    total_removed += removed;
    iframe--;
    double repro_rate = compute_R(iday, population);
    fprintf(file, "%d\t%d\t%d\t%d\t%.2f\n", iframe, iday, removed, total_removed,
	    repro_rate);
    printf("frame = %d, day = %d, new reported cases = %d, total declared cases = %d, R = %.2f\n", iframe, iday, removed, total_removed,
	   repro_rate);
    iframe++;
    iday++;
  } // while total_removed
  
  fclose(file);
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printf("\n\nExecution time = %.5f\n", time_spent);

  hipFree(population);
  hipFree(devStates);
  hipFree(num_removed);
  hipFree(num_exposed);
  hipFree(num_infected);
  hipFree(num_susceptible);
  
}
